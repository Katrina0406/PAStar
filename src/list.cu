#include "hip/hip_runtime.h"
#include "list.h"
#include "GAStar.h"
#include <assert.h>
#include <stdio.h>

llist *list_create(int capacity) {
	llist list_cpu;
	llist *list_gpu;
	list_cpu.length = 0;
	list_cpu.capacity = capacity;
	hipMalloc(&(list_cpu.arr), (capacity + 1) * sizeof(AStarNode*));
	hipMalloc(&list_gpu, sizeof(struct list));
	hipMemcpy(list_gpu, &list_cpu, sizeof(struct list),
				hipMemcpyDefault);
	return list_gpu;
}

void lists_destroy(llist **lists_gpu, int lists) {
	llist **lists_cpu = (llist**)malloc(lists * sizeof(llist*));
	hipMemcpy(lists_cpu, lists_gpu, lists * sizeof(llist*), hipMemcpyDefault);
	for (int i = 0; i < lists; i++) {
		list_destroy(lists_cpu[i]);
	}
	hipFree(lists_gpu);
	free(lists_cpu);
}

void list_destroy(llist *list_gpu) {
	llist list_cpu;
	hipMemcpy(&list_cpu, list_gpu, sizeof(struct llist),
				hipMemcpyDefault);
	hipFree(list_cpu.arr);
	hipFree(list_gpu);
}
__device__ void list_clear(llist *list) {
	list->length = 0;
}

__device__ void list_insert(llist *list, GNode *node) {
	int index = atomicAdd(&(list->length), 1);
	assert(index < llist->capacity);
	list->arr[index] = node;
}

__device__ void list_remove(llist *list, int index) {
	assert(llist->length < llist->capacity);
	list->arr[index] = NULL;
}

__device__ AStarNode *list_get(llist *list, int index) {
	assert(index < llist->length);
	return list->arr[index];
}
