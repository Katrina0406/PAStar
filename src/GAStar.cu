#include "hip/hip_runtime.h"
#include "GAStar.h"
#include "list.h"
#include "heap.h"

__global__ void clear_open_list(llist *S);
__global__ void fill_open_list(int k);
__global__ void deduplicate(llist *T);
__global__ void push_to_queues(int k, heap_open_t **open_list, llist *S, int off);

__device__ unsigned int jenkins_hash(int j, AStarNode *node);
__device__ int calculate_index();

__device__ int calculate_index() {
	return  blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int total_open_list_size = 0;
__device__ int found = 0;
__device__ int out_of_memory = 0;

void GAStar::updatePath(const LLNode* goal, vector<PathEntry> &path)
{
    const LLNode* curr = goal;
    if (curr->is_goal)
        curr = curr->parent;
    path.reserve(curr->g_val + 1);
    while (curr != nullptr)
    {
        path.emplace_back(curr->location);
        curr = curr->parent;
    }
    std::reverse(path.begin(),path.end());
}


Path GAStar::findOptimalPath()
{
    return findSuboptimalPath();
}

__device__ unsigned int jenkins_hash(int j, AStarNode *node) {
	char c;
	unsigned long hash = (j * 10000007);
	while (c = node->location++) {
		hash += c;
		hash += hash << 10;
		hash ^= hash >> 6;
	}
	hash += hash << 3;
	hash ^= hash >> 11;
	hash += hash << 15;
	return hash;
}

// find path by time-space A* search
// Returns a bounded-suboptimal path that satisfies the constraints of the give node  while
// minimizing the number of internal conflicts (that is conflicts with known_paths for other agents found so far).
// lowerbound is an underestimation of the length of the path in order to speed up the search.
Path GAStar::findSuboptimalPath()
{
    Path path;
    num_expanded = 0;
    num_generated = 0;

	int k = THREADS_PER_BLOCK * BLOCKS;


	hipMalloc(&allNodes_table, HASH_SIZE * sizeof(AStarNode*));
	hipMemset(allNodes_table, 0, HASH_SIZE * sizeof(AStarNode*));
	// priority queues of open lists (Q)
	heap_open_t **open_list = heaps_create(k);
	llist **Ss = lists_create(BLOCKS, 1000000);
	llist *S = list_create(1024 * 1024);
	int total_open_list_size_cpu;
	int found_cpu;
	int out_of_memory_cpu;

	auto start = new AStarNode(start_location, 0, compute_heuristic(start_location, goal_location), nullptr, 0, 0);
	pushNode(open_list[0], start)
	atomicAdd(&total_open_list_size, 1);
	int step = 0;

    do {
		clear_open_list<<<1, 1>>>(S);
		hipDeviceSynchronize();
		fill_open_list<<<BLOCKS, THREADS_PER_BLOCK>>>(k);
		hipMemcpyFromSymbol(&found_cpu, HIP_SYMBOL(found), sizeof(int));
		hipMemcpyFromSymbol(&out_of_memory, HIP_SYMBOL(found), sizeof(int));
		if (found_cpu) break;
		if (out_of_memory_cpu) break;
		hipDeviceSynchronize();
		deduplicate<<<BLOCKS, THREADS_PER_BLOCK>>>(S);
		hipDeviceSynchronize();
		push_to_queues<<<1, THREADS_PER_BLOCK>>>(k, step) ;
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&total_open_list_size_cpu, HIP_SYMBOL(total_open_list_size), sizeof(int));
		step++;
	} while (total_open_list_size_cpu > 0);


	// lists_destroy(Ss, BLOCKS);
	// heaps_destroy(open_list, k);
	// HANDLE_RESULT(hipFree(allNodes_table));
	hipDeviceSynchronize();

    // // generate start and add it to the OPEN & FOCAL list
    // auto start = new AStarNode(start_location, 0, compute_heuristic(start_location, goal_location), nullptr, 0, 0);

    // pushNode(start);
    // allNodes_table.insert(start);
    // min_f_val = (int) start->getFVal();
    // lower_bound = int(w * min_f_val));
	releaseNodes();

	return path;

}

void GAStar::expandNode(AStarNode *next, heap_open_t *open_list, llist S){
	auto next_locations = instance.getNeighbors(curr->location);
	next_locations.emplace_back(curr->location);
	for (int next_location : next_locations)
	{
		int next_timestep = curr->timestep + 1;
		// compute cost to next_id via curr node
		int next_g_val = curr->g_val + 1;
		int next_h_val = compute_heuristic(next_location, goal_location);
		
		// generate (maybe temporary) node
		auto next = new AStarNode(next_location, next_g_val, next_h_val,
									curr, next_timestep);

		list_insert(S, next);
		// delete(next);  // not needed anymore -- we already generated it before
	}
}


inline AStarNode* GAStar::popNode(heap_open_t *open_list)
{
    auto node = open_list.top(); open_list.pop();
    // open_list.erase(node->open_handle);
    node->in_openlist = false;
    num_expanded++;
    return node;
}


inline void GAStar::pushNode(heap_open_t *open_list, AStarNode* node)
{
    node->open_handle = open_list.push(node);
    node->in_openlist = true;
    num_generated++;
}

void GAStar::releaseNodes()
{
	// TODO: modify
    // open_list.clear();
    for (auto node: allNodes_table)
        delete node;
    allNodes_table.clear();
}

__global__ void clear_open_list(llist *S) {
	list_clear(S);
}

__global__ void fill_open_list(int k) {
	auto *bestNode = NULL;
	int index = calculate_index();
	if (index == 0) steps++;

	for (int i = index; i < k; i += blockDim.x * gridDim.x) {
		if (open_list[i].empty()) continue;
		auto* curr = popNode(open_list[i]);
		atomicSub(&total_open_list_size, 1);
		if (curr->location == goal_location) {
			if (bestNode == NULL || curr->getFVal() < bestNode->getFVal()) {
                // Found a better possible path starting with curr node
				bestNode = copy(*curr);
			}
			// If already at goal location, no need to expand
			continue;
		}
		// Expand S
		expandNode(curr, open_list[i]);
	}
	if (bestNode != NULL && bestNode->getFVal() <= heaps_min(open_list, k)) {
        // Found a better path, update found and return the path
		int found_before = atomicCAS(&found, 0, 1);
		if (found_before == 1) return;
		updatePath(bestNode, path);
	}
}

__global__ void deduplicate(llist *T) {
	int id = calculate_index();
	for (int i = id; i < T->length; i += blockDim.x * gridDim.x) {
		int z = 0;
		AStarNode *t1 = list_get(T, i);
		for (int j = 0; j < HASH_FUNS; j++) {
			assert(t1 != NULL);
			auto el = allNodes_table[jenkins_hash(j, t1) % HASH_SIZE];
			if (el == NULL || cuda_str_eq(t1, el)) {
				z = j;
				break;
			}
		}
		int index = jenkins_hash(z, t1) % HASH_SIZE;
		t1 = (AStarNode*)atomicExch((unsigned long long*)&(allNodes_table[jenkins_hash(z, t1) % HASH_SIZE]), (unsigned long long)t1);
		if (t1 != NULL && t1 == list_get(T, i) &&
				(list_get(T, i), t, h)->getFVal() >= t1->getFVal()) {
			list_remove(T, i);
			continue;
		}
		t1 = list_get(T, i);
		for (int j = 0; j < HASH_FUNS; j++) {
			if (j != z) {
				auto el = allNodes_table[jenkins_hash(j, t1) % HASH_SIZE];
				if (el != NULL && el == t1 &&
						(list_get(T, i), t, h)->getFVal() >= el->getFVal()) {
					list_remove(T, i);
					break;
				}
			}
		}
	}
}

__global__ void push_to_queues(int k, heap_open_t **open_list, llist *S, int off) {
	for (int i = threadIdx.x; i < S->length; i += blockDim.x) {
		AStarNode *t1 = list_get(S, i);
		if (t1 != NULL) {
			pushNode(open_list[(i + off) % k], t1);
			open_list.increase(t1->open_handle); 
			atomicAdd(&processed, 1);
			atomicAdd(&total_open_list_size, 1);
		}
		__syncthreads();
	}
}